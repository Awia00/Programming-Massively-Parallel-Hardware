#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#if defined(_OPENMP)
#include "omp.h"
#else
#include "Host.cu.h"
#endif

#if defined(WIN32) || defined(_WIN32) || defined(__WIN32)
#include "TimeOfDay.h" // Only on windows
#else
#include <sys/time.h> // Only on linux
#include "Main.h"
#endif




int timeval_subtract(struct timeval *result, struct timeval *t2, struct timeval *t1)
{
	unsigned int resolution = 1000000;
	long int diff = (t2->tv_usec + resolution * t2->tv_sec) - (t1->tv_usec + resolution * t1->tv_sec);
	result->tv_sec = diff / resolution;
	result->tv_usec = diff % resolution;
	return (diff<0);
}

#pragma region MatrixTranspose  

void matrixTransposeSequential(float* matrix, float* outMatrix, int M, int N) {
	for(int i = 0; i < M; i++) {
		for(int j = 0; j < N; j++){
			outMatrix[j*M + i] = matrix[i*N + j];
		}
	}
}

#if defined(_OPENMP)

void matrixTransposeOMP(float* matrix, float* outMatrix, int M, int N) {
#pragma omp parallel shared(matrix, outMatrix, M, N) default(none)
	for (int i = 0; i < M; i++) {
		for (int j = 0; j < N; j++) {
			outMatrix[j*M + i] = matrix[i*N + j];
		}
	}
}


void matrixTransposeOMPTest() {
	printf("\nRunning Matrix Transpose OMP\n");

	const unsigned int N = 8000;
	const unsigned int M = 8000;
	unsigned int mem_size_A = M*N * sizeof(float);
	float* h_A = (float*)malloc(mem_size_A);
	float* h_A_out = (float*)malloc(mem_size_A);
	float* h_A_expected = (float*)malloc(mem_size_A);

	for (int i = 0; i<M; i++) {
		for (int j = 0; j<N; j++) {
			h_A[i*N + j] = rand() % 100;
		}
	}

	matrixTransposeOMP(h_A, h_A_out, M, N);
	matrixTransposeSequential(h_A, h_A_expected, M, N);

	bool succeded;
	for (int i = 0; i<M; i++) {
		for (int j = 0; j<N; j++) {
			if (h_A_out[i*N + j] != h_A_expected[i*N + j]) {
				printf("\nFailed. Expected %f, was %f\n", h_A_expected[i*N + j], h_A_out[i*N + j]);
				succeded = false;
				break;
			}
		}
		if (!succeded)
			break;
	}
	if (succeded)
		printf("\nCompleted successfully\n");

	free(h_A);
	free(h_A_out);
}

#else

void matrixTransposeGPU(unsigned int mem_size_A, float * h_A, float * h_out, const unsigned int &M, const unsigned int &N, bool optimized)
{
	const unsigned int block_size = 512;
	const unsigned int T = 64;

	float* d_A;
	float* d_out;
	hipMalloc((void**)&d_A, mem_size_A);
	hipMalloc((void**)&d_out, mem_size_A);
	// copy host memory to device
	hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);

	unsigned long int elapsed;
	struct timeval t_start, t_end, t_diff;
	gettimeofday(&t_start, NULL);

	// run 
	matrixTranspose<float, T>(block_size, d_A, d_out, M, N, optimized);
	
	hipMemcpy(h_out, d_out, mem_size_A, hipMemcpyDeviceToHost);

	gettimeofday(&t_end, NULL);
	timeval_subtract(&t_diff, &t_end, &t_start);
	elapsed = (t_diff.tv_sec*1e6 + t_diff.tv_usec);

	hipFree(d_A);
	hipFree(d_out);

	// print results
	if (optimized) {
		printf("M-transpose optimized time\t %d\n", elapsed);
	}
	else {
		printf("M-transpose naive time\t\t %d\n", elapsed);
	}
}

void matrixTransposeGPUTest(bool optimized) {
	if(optimized)
		printf("\nRunning Matrix Transpose optimized\n");
	else
		printf("\nRunning Matrix Transpose naive\n");

	const unsigned int N = 5000;
	const unsigned int M = 4000;
	unsigned int mem_size_A = M*N * sizeof(float);	
	float* h_A = (float*)malloc(mem_size_A);
	float* h_A_out = (float*)malloc(mem_size_A);
	float* h_A_expected = (float*)malloc(mem_size_A);

	for(int i = 0; i<M; i++) {
		for(int j = 0; j<N; j++){
			h_A[i*N + j] = rand() % 100;
		}
	}

	matrixTransposeGPU(mem_size_A, h_A, h_A_out, M, N, optimized);
	matrixTransposeSequential(h_A, h_A_expected, M, N);

	bool succeded;
	for (int i = 0; i<M; i++) {
		for (int j = 0; j<N; j++) {
			if (h_A_out[i*N + j] != h_A_expected[i*N + j]) {
				printf("\nFailed. Expected %f, was %f\n", h_A_expected[i*N + j], h_A_out[i*N + j]);
				succeded = false;
				break;
			}
		}
		if (!succeded)
			break;
	}
	if (succeded)
		printf("\nCompleted successfully\n");

	free(h_A);
	free(h_A_out);
}



#endif

#pragma endregion MatrixTranspose

#pragma region squareAccumulator

void squareAccumulatorSequential(float* A, float* B, int N) {
	int M = 64;
	for (int i = 0; i < N; i++) {
		float accum = A[i*M];
		B[i*M] = accum;
		for (int j = 1; j < 64; j++) {
			float tmpA = A[i*M + j];
			accum = sqrt(accum) + tmpA*tmpA;
			B[i*M + j] = accum;
		}
	}
}

#if defined(_OPENMP)

void squareAccumulatorOMP(float* A, float* B, int N) {
	const int M = 64;
	#pragma omp parallel shared(A,B,N) default(none)
	for (int i = 0; i < N; i++) {
		float accum = A[i*M];
		B[i*M] = accum;
		for (int j = 1; j < 64; j++) {
			float tmpA = A[i*M + j];
			accum = sqrt(accum) + tmpA*tmpA;
			B[i*M + j] = accum;
		}
	}
}

void squareAccumulatorOMPTest() {
	printf("\nRunning Square Accumulator optimized\n");

	const unsigned int T = 64;
	const unsigned int N = 200000;
	unsigned int mem_size = N*T * sizeof(float);
	float* h_A = (float*)malloc(mem_size);
	float* h_B = (float*)malloc(mem_size);
	float* h_B_expected = (float*)malloc(mem_size);

	for (int i = 0; i<N; i++) {
		for (int j = 0; j<T; j++) {
			h_A[i*T + j] = rand() % 100;
		}
	}

	squareAccumulatorOMP(h_A, h_B, N);
	squareAccumulatorSequential(h_A, h_B_expected, N);

	bool succeded;
	for (int i = 0; i<N; i++) {
		for (int j = 0; j<T; j++) {
			if (h_B[i*N + j] != h_B_expected[i*N + j]) {
				printf("\nFailed. Expected %f, was %f\n", h_B_expected[i*N + j], h_B[i*N + j]);
				succeded = false;
				break;
			}
		}
		if (!succeded)
			break;
	}
	if (succeded)
		printf("\nCompleted successfully\n");

	free(h_A);
	free(h_B);
}

#else

void squareAccumulatorGPU(unsigned int mem_size, float * h_A, float * h_B, const unsigned int &N, bool optimized)
{
	float* d_A;
	float* d_B;
	const unsigned int block_size = 512;
	const unsigned int T = 64;
	hipMalloc((void**)&d_A, mem_size);
	hipMalloc((void**)&d_B, mem_size);
	// copy host memory to device
	hipMemcpy(d_A, h_A, mem_size, hipMemcpyHostToDevice);

	unsigned long int elapsed;
	struct timeval t_start, t_end, t_diff;
	gettimeofday(&t_start, NULL);

	// run 
	squareAccumulator<T>(block_size, N, d_A, d_B, optimized);
	hipDeviceSynchronize();

	gettimeofday(&t_end, NULL);
	timeval_subtract(&t_diff, &t_end, &t_start);
	elapsed = (t_diff.tv_sec*1e6 + t_diff.tv_usec);

	hipMemcpy(h_B, d_B, mem_size, hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);

	// print results
	if (optimized)
		printf("Square Accumulator optimized time\t %d\n", elapsed);
	else
		printf("Square Accumulator naive time\t\t %d\n", elapsed);
}

void squareAccumulatorGPUTest(bool optimized) {
	if (optimized)
		printf("\nRunning Square Accumulator optimized\n");
	else
		printf("\nRunning Square Accumulator naive\n");

	const unsigned int T = 64;
	const unsigned int N = 200000;
	unsigned int mem_size = N*T * sizeof(float);	
	float* h_A = (float*)malloc(mem_size);
	float* h_B = (float*)malloc(mem_size);
	float* h_B_expected = (float*)malloc(mem_size);

	for(int i = 0; i<N; i++) {
		for(int j = 0; j<T; j++){
			h_A[i*T + j] = rand() % 100;
		}
	}

	squareAccumulatorGPU(mem_size, h_A, h_B, N, optimized);
	squareAccumulatorSequential(h_A, h_B_expected, N);

	bool succeded;
	for (int i = 0; i<N; i++) {
		for (int j = 0; j<T; j++) {
			if (h_B[i*N + j] != h_B_expected[i*N + j]) {
				printf("\nFailed. Expected %f, was %f\n", h_B_expected[i*N + j], h_B[i*N + j]);
				succeded = false;
				break;
			}
		}
		if (!succeded)
			break;
	}
	if (succeded)
		printf("\nCompleted successfully\n");

	free(h_A);
	free(h_B);
}

#endif

#pragma endregion SquareAccumulator

#pragma region MatrixMatrixMul

void matrixMatrixMulSequential(float* A, float* B, float* C, int N, int M, int U) {
	for(int i = 0; i < M; i++) {
		for(int j = 0; j < N; j++) {
			float tmp = 0.0f;
			for(int k = 0; k < U; k++)
				tmp += A[i*U + k] * B[k*N + j];
			C[i*N + j] = tmp;
		}
	}
}

#if defined(_OPENMP)

void matrixMatrixMulOMP(float* A, float* B, float* C, int N, int M, int U) {
#pragma omp parallel shared(A,B,C,N,M,U) default(none)
	for (int i = 0; i < M; i++) {
		for (int j = 0; j < N; j++) {
			float tmp = 0.0f;
			for (int k = 0; k < U; k++)
				tmp += A[i*U + k] * B[k*N + j];
			C[i*N + j] = tmp;
		}
	}
}
void matrixMatrixMulOMPTest() {
	printf("\nRunning Matrix Matrix Mul OMP\n");

	const unsigned int N = 3000;
	const unsigned int M = 3000;
	const unsigned int U = 3000;
	unsigned int mem_size_A = M*U * sizeof(float);
	unsigned int mem_size_B = U*N * sizeof(float);
	unsigned int mem_size_C = M*N * sizeof(float);
	float* h_A = (float*)malloc(mem_size_A);
	float* h_B = (float*)malloc(mem_size_B);
	float* h_C = (float*)malloc(mem_size_C);
	float* h_C_expected = (float*)malloc(mem_size_C);

	for (int i = 0; i<M; i++) {
		for (int j = 0; j<U; j++) {
			h_A[i*U + j] = rand() % 100;
		}
	}
	for (int i = 0; i<U; i++) {
		for (int j = 0; j<N; j++) {
			h_B[i*N + j] = rand() % 100;
		}
	}

	matrixMatrixMulOMP(h_A, h_B, h_C, M, N, U);
	matrixMatrixMulSequential(h_A, h_B, h_C, N, M, U);

	bool succeded;
	for (int i = 0; i<M; i++) {
		for (int j = 0; j<N; j++) {
			if (abs(h_C[i*N + j] - h_C_expected[i*N + j]) < 0.1f) {
				printf("\nFailed. Expected %f, was %f\n", h_C[i*N + j], h_C[i*N + j]);
				succeded = false;
				break;
			}
		}
		if (!succeded)
			break;
	}
	if (succeded)
		printf("\nCompleted successfully\n");


	free(h_A);
	free(h_B);
	free(h_C);
}

#else

void matrixMatrixMulGPU(
	unsigned int mem_size_A, 
	unsigned int mem_size_B, 
	unsigned int mem_size_C,
	float * h_A, 
	float * h_B, 
	float * h_C, 
	const unsigned int &M, 
	const unsigned int &N, 
	const unsigned int &U, 
	bool optimized)
{
	const unsigned int block_size = 512;
	const unsigned int T = 64;
	float* d_A;
	float* d_B;
	float* d_C;
	hipMalloc((void**)&d_A, mem_size_A);
	hipMalloc((void**)&d_B, mem_size_B);
	hipMalloc((void**)&d_C, mem_size_C);
	// copy host memory to device
	hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

	unsigned long int elapsed;
	struct timeval t_start, t_end, t_diff;
	gettimeofday(&t_start, NULL);

	// run 
	matrixMatrixMul<T>(d_A, d_B, d_C, M, N, U, block_size, optimized);
	hipDeviceSynchronize();

	gettimeofday(&t_end, NULL);
	timeval_subtract(&t_diff, &t_end, &t_start);
	elapsed = (t_diff.tv_sec*1e6 + t_diff.tv_usec);
	double flops = 2.0 * M * N * U;
	double gigaFlops = (flops*1.0e-3f) / elapsed;

	hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// Print the results
	if (optimized) {
		printf("MMM optimized time\t\t %d\n", elapsed);
		printf("MMM gigaFlops optimized\t %f\n", gigaFlops);
	}
	else {
		printf("MMM naive time\t\t %d\n", elapsed);
		printf("MMM gigaFlops naive\t %f\n", gigaFlops);
	}
}

//  A, B and C have sizes MxU, UxN and MxN
void matrixMatrixMulGPUTest(bool optimized){
	if (optimized)
		printf("\nRunning Matrix Matrix Mul optimized\n");
	else
		printf("\nRunning Matrix Matrix Mul naive\n");

	const unsigned int N = 2500;
	const unsigned int M = 2500;
	const unsigned int U = 2500;
	unsigned int mem_size_A = M*U * sizeof(float);	
	unsigned int mem_size_B = U*N * sizeof(float);	
	unsigned int mem_size_C = M*N * sizeof(float);
	float* h_A = (float*)malloc(mem_size_A);
	float* h_B = (float*)malloc(mem_size_B);
	float* h_C = (float*)malloc(mem_size_C);
	float* h_C_expected = (float*)malloc(mem_size_C);

	for(int i = 0; i<M; i++) {
		for(int j = 0; j<U; j++){
			h_A[i*U + j] = rand() % 100;
		}
	}
	for(int i = 0; i<U; i++) {
		for(int j = 0; j<N; j++){
			h_B[i*N + j] = rand() % 100;
		}
	}

	matrixMatrixMulGPU(mem_size_A, mem_size_B, mem_size_C, h_A, h_B, h_C, M, N, U, optimized);
	matrixMatrixMulSequential(h_A, h_B, h_C, N, M, U);

	bool succeded;
	for (int i = 0; i<M; i++) {
		for (int j = 0; j<N; j++) {
			if (abs(h_C[i*N + j] - h_C_expected[i*N + j]) < 0.1f) {
				printf("\nFailed. Expected %f, was %f\n", h_C[i*N + j], h_C[i*N + j]);
				succeded = false;
				break;
			}
		}
		if (!succeded)
			break;
	}
	if (succeded)
		printf("\nCompleted successfully\n");
	

	free(h_A);
	free(h_B);
	free(h_C);
}

#endif

#pragma endregion MatrixMatrixMul


// ====== Runners

#if defined(_OPENMP)

void runOpenMPProgram() {
	matrixTransposeOMPTest();
	printf("\n==========================\n");
	matrixTransposeOMPTest();
}

#else 

void runGPUProgram() {
	matrixTransposeGPUTest(false);
	printf("\n==========================\n");
	matrixTransposeGPUTest(true);

	printf("\n==========================");
	printf("\n==========================\n");

	squareAccumulatorGPUTest(false);
	printf("\n==========================\n");
	squareAccumulatorGPUTest(true);

	printf("\n==========================");
	printf("\n==========================\n");

	matrixMatrixMulGPUTest(false);
	printf("\n==========================\n");
	matrixMatrixMulGPUTest(true);
}

#endif



int main(int argc, char** argv) {
#if defined(_OPENMP)
	runOpenMPProgram();
#else
	runGPUProgram();
#endif
}
