#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "Host.cu.h"

#if defined(WIN32) || defined(_WIN32) || defined(__WIN32) && !defined(__CYGWIN__)
#include "TimeOfDay.h" // Only on windows
#else
#include <sys/time.h> // Only on linux
#endif


void spMatrixVctMultiply(float* mat_val, int* mat_inds, float* vct, int num_rows, int* shp, float* res) {
	int offset = 0;
	for (int i = 0; i < num_rows; i++) {
		int sum = 0, row_len = shp[i];
		for (int j = 0; j < row_len; j++) {
			sum += mat_val[j + offset] * vct[mat_inds[j + offset]];
		}
		offset += row_len;
		res[i] = sum;
	}
}

void spMatrixVctTest() {
	const unsigned int block_size = 512;
	const unsigned int matrix_height = 5733;
	const unsigned int matrix_width = 5733;
	const unsigned int matrix_size = matrix_height * matrix_width;
	const unsigned int vector_size = matrix_height;

	unsigned int mem_size_mat_vals	= matrix_size * sizeof(float);
	unsigned int mem_size_mat_inds	= matrix_size * sizeof(int);
	unsigned int mem_size_shp		= matrix_height * sizeof(int);
	unsigned int mem_size_vct		= vector_size * sizeof(float);

	float* h_mat_val = (float*)malloc(mem_size_mat_vals);
	float* h_vct = (float*)malloc(mem_size_vct);
	float* h_out = (float*)malloc(mem_size_vct);
	int* h_mat_inds = (int*)calloc(matrix_size, sizeof(int));
	int* h_shp = (int*)calloc(matrix_height, sizeof(int));
	int* h_flags = (int*)calloc(matrix_size, sizeof(int));

	// generate matrix. Did not generate a matrix with any sparse entries, but code shuld be able to handle it.
	for (unsigned int i = 0; i < matrix_height; i++) {
		int nonZeros = 0;
		for (unsigned int j = 0; j < matrix_width; j++) {
			h_mat_val[i*matrix_height + j] = rand() % 100 + 1; // 1-100
			h_mat_inds[i*matrix_height + j] = j; // 1-100
			nonZeros++;
		}
		h_shp[i] = nonZeros;
		h_vct[i] = rand() % 100 + 1;
		h_flags[i*matrix_width] = 1;
	}

	unsigned long int elapsed, elapsedCPU;
	struct timeval t_start, t_end, t_diff;
	gettimeofday(&t_start, NULL);

	{ // calling exclusive (segmented) scan
		float* d_mat_val;
		float* d_vct;
		float* d_out;
		int* d_mat_inds;
		int* d_flags;
		
		hipMalloc((void**)&d_mat_val, mem_size_mat_vals);
		hipMalloc((void**)&d_vct, mem_size_vct);
		hipMalloc((void**)&d_out, mem_size_vct);
		hipMalloc((void**)&d_mat_inds, mem_size_mat_inds);
		hipMalloc((void**)&d_flags, mem_size_mat_inds);

		// copy host memory to device
		hipMemcpy(d_mat_val, h_mat_val, mem_size_mat_vals, hipMemcpyHostToDevice);
		hipMemcpy(d_vct, h_vct, mem_size_vct, hipMemcpyHostToDevice);
		hipMemcpy(d_mat_inds, h_mat_inds, mem_size_mat_inds, hipMemcpyHostToDevice);
		hipMemcpy(d_flags, h_flags, mem_size_mat_inds, hipMemcpyHostToDevice);

		// execute kernel
		sp_matrix_vector_multiply(block_size, matrix_size, vector_size, d_mat_inds, d_mat_val, d_vct, d_flags, d_out);

		// copy host memory to device
		hipMemcpy(h_out, d_out, mem_size_vct, hipMemcpyDeviceToHost);

		// cleanup memory
		hipFree(d_mat_val);
		hipFree(d_vct);
		hipFree(d_mat_inds);
		hipFree(d_flags);
		hipFree(d_out);
	}

	gettimeofday(&t_end, NULL);
	timeval_subtract(&t_diff, &t_end, &t_start);
	elapsed = (t_diff.tv_sec*1e6 + t_diff.tv_usec);
	printf("SP MV MUL GPU Kernel runs in: \t%lu microsecs\n", elapsed);

	// validate
	float* h_test = (float*)malloc(mem_size_vct);
	
	gettimeofday(&t_start, NULL);
	spMatrixVctMultiply(h_mat_val, h_mat_inds, h_vct, matrix_height, h_shp, h_test);
	gettimeofday(&t_end, NULL);
	timeval_subtract(&t_diff, &t_end, &t_start);
	elapsedCPU = (t_diff.tv_sec*1e6 + t_diff.tv_usec);
	printf("SP MV MUL CPU runs in: \t\t%lu microsecs\n", elapsedCPU);

	elapsed = (t_diff.tv_sec*1e6 + t_diff.tv_usec);


	bool success = true;
	for (int i = 0; i < matrix_height; i++) {
		if (abs(h_test[i] - h_out[i]) > 0.0001f) {
			success = false;
			printf("Failed at: h_test[%d]: %f, h_out[%d]=%f\n", i, h_test[i], i, h_out[i]);
			break;
		}
	}

	if (success) printf("Sparse Matrix Vector Multiply +   VALID RESULT!\n");
	else        printf("Sparse Matrix Vector Multiply + INVALID RESULT!\n");

	free(h_mat_val);
	free(h_vct);
	free(h_out);
	free(h_mat_inds);
	free(h_flags);
}

void matrixTranspose(float* matrix, float* outMatrix, int n, int m) {
	for(int i = 0; i < m; i++) {
		for(int j = 0; j < n; j++){
			outMatrix[j][i] = matrix[i][j];
		}
	}
}

void matrixTransposeTest(bool optimized) {
	const unsigned int block_size = 512;
	const unsigned int N = 1000;
	const unsigned int M = 1000;
	unsigned int mem_size_A = M*N * sizeof(float);	
	float* h_A = (float*)malloc(mem_size_A);
	float* h_A_out = (float*)malloc(mem_size_A);

	for(int i = 0; i<M; i++) {
		for(int j = 0; j<N; j++){
			A[i*M + j] = rand() % 100;
		}
	}
	
	unsigned long int elapsed;
	struct timeval t_start, t_end, t_diff;

	float* d_A;
	float* d_A_out;
	hipMalloc((void**)&d_A, mem_size_A);
	hipMalloc((void**)&d_A_out, mem_size_A);
	// copy host memory to device
	hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
	hipMemcpy(d_A_out, h_A_out, mem_size_A, hipMemcpyHostToDevice);

	unsigned long int elapsed;
	struct timeval t_start,t_end,t_diff;
	gettimeofday(&t_start, NULL);
	
	// run 
	matrix_transpose(d_A, d_A_out, M, N, optimized);
	__syncThreads();
	
	gettimeofday(&t_end, NULL);
	timeval_subtract(&t_diff, &t_end,&t_start);
	if(optimized) {
		printf("M-transpose optimized time\t %d", elapsedTime);
	} else { 
		printf("M-transpose naive time\t %d", elapsedTime);
	}

	hipMemcpy(h_A_out, d_A_out, mem_size_A, hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_A_out);
}

void squareAccumulatorTest(bool optimized) {
	const unsigned int block_size = 512;
	const unsigned int N = 1000;
	const unsigned int M = 64;
	unsigned int mem_size = M*N * sizeof(float);	
	float* h_A = (float*)malloc(mem_size);
	float* h_B = (float*)malloc(mem_size);

	for(int i = 0; i<M; i++) {
		for(int j = 0; j<N; j++){
			A[i*M + j] = rand() % 100;
		}
	}
	
	unsigned long int elapsed;
	struct timeval t_start, t_end, t_diff;

	float* d_A;
	float* d_B;
	hipMalloc((void**)&d_A, mem_size);
	hipMalloc((void**)&d_B, mem_size);
	// copy host memory to device
	hipMemcpy(d_A, h_A, mem_size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, mem_size, hipMemcpyHostToDevice);

	unsigned long int elapsed;
	struct timeval t_start,t_end,t_diff;
	gettimeofday(&t_start, NULL);
	
	// run 
	square_accumulator(N, d_A, d_B, optimized);
	__syncThreads();
	
	gettimeofday(&t_end, NULL);
	timeval_subtract(&t_diff, &t_end,&t_start);

	if(optimized)
		printf("Square Accumulator optimized time\t %d", elapsedTime);	
	else
		printf("Square Accumulator naive time\t %d", elapsedTime);	
	
	hipMemcpy(h_B, d_B, mem_size, hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
}

void denseMatrixMatrixMul(float* A, float* B, float* C, int N, int M, int U) {
	for(int i = 0; i < M; i++) {
		for(int j = 0; j < N; j++) {
			float tmp = 0.0f;
			for(int k = 0; k < U; k++)
				tmp += A[i*M + k] * B[k*U + j];
			C[i*M + j] = tmp;
		}
	}
}

//  A, B and C have sizes MxU, UxN and MxN
void matrixMatrixMulTest(bool optimized){
	const unsigned int block_size = 512;
	const unsigned int N = 1000;
	const unsigned int M = 1000;
	const unsigned int U = 1000;
	unsigned int mem_size_A = M*U * sizeof(float);	
	unsigned int mem_size_B = U*N * sizeof(float);	
	unsigned int mem_size_C = M*N * sizeof(float);
	float* h_A = (float*)malloc(mem_size_A);
	float* h_B = (float*)malloc(mem_size_B);
	float* h_C = (float*)malloc(mem_size_C);

	for(int i = 0; i<M; i++) {
		for(int j = 0; j<U; j++){
			A[i*M + j] = rand() % 100;
		}
	}
	for(int i = 0; i<U; i++) {
		for(int j = 0; j<N; j++){
			B[i*U + j] = rand() % 100;
		}
	}
	for(int i = 0; i<M; i++) {
		for(int j = 0; j<N; j++){
			C[i*M + j] = rand() % 100;
		}
	}
	
	unsigned long int elapsed;
	struct timeval t_start, t_end, t_diff;

	float* d_A;
	float* d_B;
	float* d_C;
	hipMalloc((void**)&d_A, mem_size_A);
	hipMalloc((void**)&d_B, mem_size_B);
	hipMalloc((void**)&d_C, mem_size_C);
	// copy host memory to device
	hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, mem_size_C, hipMemcpyHostToDevice);


	unsigned long int elapsed;
	struct timeval t_start,t_end,t_diff;
	gettimeofday(&t_start, NULL);
	
	// run 
	matrix_matrix_mul(d_A, d_B, d_C, M, N, U, optimized);
	__syncThreads();
	
	gettimeofday(&t_end, NULL);
	timeval_subtract(&t_diff, &t_end,&t_start);
	elapsed=(t_diff.tv_sec*1e6 + t_diff.tv_usec);
	double flops = 2.0 * M * N * U;
	double gigaFlops=(flops*1.0e-3f) /elapsed;
	if(optimized) {
		printf("MMM optimized time\t %d", elapsedTime);		
		printf("MMM gigaFlops optimized %d", gigaFlops);
	} else { 
		printf("MMM naive time\t %d", elapsedTime);				
		printf("MMM gigaFlops naive %d", gigaFlops);
	}

	hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

float gflops(){
	unsigned long int elapsed;
	struct timeval t_start,t_end,t_diff;
	gettimeofday(&t_start, NULL);
	
	gettimeofday(&t_end, NULL);
	timeval_subtract(&t_diff, &t_end,&t_start);
	elapsed=(t_diff.tv_sec*1e6 + t_diff.tv_usec);
	double flops = 2.0 * M * N * U;
	double gigaFlops=(flops*1.0e-3f) /elapsed;
}
int main(int argc, char** argv) {
	printf("\n");

	spMatrixVctTest();
	printf("\n==========================\n");
	spMatrixVctTest();
}

